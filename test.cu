#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

// CUDA内核函数，用于生成随机数

void fuck() { int a = 1; }
__global__ void randomKernel(float* randomArray, int arraySize) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < arraySize) {
    printf("32\n");
    hiprandState_t state;
    hiprand_init(0, tid, 0, &state);  // 使用0作为种子值，tid是线程ID
    fuck();
    // 生成随机数
    randomArray[tid] = hiprand_uniform(&state);
  }
}

int main() {
  int arraySize = 1000;
  float randomArray[arraySize];

  // 分配和初始化设备数组
  float* devRandomArray;
  hipMalloc((void**)&devRandomArray, arraySize * sizeof(float));
  hipMemcpy(devRandomArray, randomArray, arraySize * sizeof(float),
             hipMemcpyHostToDevice);

  // 启动内核函数
  int blockSize = 512;
  int gridSize = (arraySize + blockSize - 1) / blockSize;
  randomKernel<<<gridSize, blockSize>>>(devRandomArray, arraySize);

  // 将结果从设备复制回主机
  hipMemcpy(randomArray, devRandomArray, arraySize * sizeof(float),
             hipMemcpyDeviceToHost);

  // 打印随机数

  // 清理资源
  hipFree(devRandomArray);

  return 0;
}
